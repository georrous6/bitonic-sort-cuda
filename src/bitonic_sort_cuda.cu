#include "bitonic_sort_cuda.cuh"
#include "util.cuh"
#include "kernel.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "config.cuh"


__host__
int wakeup_cuda(void) {
    wakeup_kernel<<<1, 1>>>();
    return post_launch_barrier_and_check();
}


__host__
static void bitonic_sort_serial(int *data, int n, int ascending) {
    for (int size = 2; size <= n; size <<= 1) {
        for (int step = size >> 1; step > 0; step >>= 1) {
            for (int i = 0; i < n; i++) {
                int j = i ^ step;
                if (j > i) {
                    int is_ascending = ((i & size) == 0) ? ascending : !ascending;
                    compare_and_swap(data, i, j, is_ascending);
                }
            }
        }
    }
}


__host__
static int bitonic_sort_v0(int *host_data, int n, int ascending) {

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int *device_data = NULL;

    if (host_to_device_data(host_data, n, &device_data) != EXIT_SUCCESS) {
        return EXIT_FAILURE;
    }

    for (int size = 2; size <= n; size <<= 1) {
        for (int step = size >> 1; step > 0; step >>= 1) {

            // Launch the kernel
            kernel_v0<<<numBlocks, BLOCK_SIZE>>>(device_data, n, ascending, size, step);

            if (post_launch_barrier_and_check()) {
                hipFree(device_data);
                return EXIT_FAILURE;
            }
        }
    }

    // Copy the sorted data back to host
    if (device_to_host_data(host_data, n, device_data) != EXIT_SUCCESS) {
        hipFree(device_data);
        return EXIT_FAILURE;
    }

    hipFree(device_data);
    return EXIT_SUCCESS;
}


__host__
static int bitonic_sort_v1(int *host_data, int n, int ascending) {
    // For simplicity, require n to be divisible by BLOCK_SIZE * numBlocks
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int chunk_size   = n / numBlocks;
    int max_step  = chunk_size >> 1;  // half block size

    int *device_data = nullptr;
    if (host_to_device_data(host_data, n, &device_data) != EXIT_SUCCESS)
        return EXIT_FAILURE;

    // 1) initial alternating sort in each block
    kernel_v1_alternating_sort<<<numBlocks, BLOCK_SIZE>>>(device_data, chunk_size, ascending);
    if (post_launch_barrier_and_check()) {
        hipFree(device_data);
        return EXIT_FAILURE;
    }

    // 2) merge across blocks
    for (int size = chunk_size << 1; size <= n; size <<= 1) {
        // global compare steps from size/2 down to chunk_size/2
        for (int step = size >> 1; step >= max_step; step >>= 1) {
            kernel_v0<<<numBlocks, BLOCK_SIZE>>>(device_data, n, ascending, size, step);
            if (post_launch_barrier_and_check()) {
                hipFree(device_data);
                return EXIT_FAILURE;
            }
        }
        // 3) intra-block refine for next bitonic run
        kernel_v1_intra_block_sort<<<numBlocks, BLOCK_SIZE>>>(device_data,
                                                              chunk_size,
                                                              ascending,
                                                              size,
                                                              max_step);
        if (post_launch_barrier_and_check()) {
            hipFree(device_data);
            return EXIT_FAILURE;
        }
    }

    if (device_to_host_data(host_data, n, device_data) != EXIT_SUCCESS) {
        hipFree(device_data);
        return EXIT_FAILURE;
    }

    hipFree(device_data);
    return EXIT_SUCCESS;
}


__host__
static int bitonic_sort_v2(int *host_data, int n, int ascending) {
    // Placeholder for future kernel version 2 implementation
    fprintf(stderr, "Kernel version 2 is not implemented yet.\n");
    return EXIT_FAILURE;
}


__host__
int bitonic_sort_cuda(int *data, int n, int ascending, kernel_version_t kernel_version) {

    if ((n & (n - 1)) != 0) {
        fprintf(stderr, "Error: Input size n=%d is not a power of 2.\n", n);
        return EXIT_FAILURE;
    }


    int status = EXIT_SUCCESS;
    // Kernel launch
    switch (kernel_version) {
        case KERNEL_NONE:
            bitonic_sort_serial(data, n, ascending);
            break;
        case KERNEL_V0:
            status = bitonic_sort_v0(data, n, ascending);
            break;
        case KERNEL_V1:
            status = bitonic_sort_v1(data, n, ascending);
            break;
        case KERNEL_V2:
            status = bitonic_sort_v2(data, n, ascending);
            break;
        default:
            fprintf(stderr, "Unsupported kernel version: %d\n", kernel_version);
            status = EXIT_FAILURE;
    }

    if (status) {
        fprintf(stderr, "Fallback to serial bitonic sort.\n");
        bitonic_sort_serial(data, n, ascending);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
