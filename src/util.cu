#include "hip/hip_runtime.h"
#include "util.cuh"
#include <stdio.h>


namespace util {

__host__
int host_to_device_data(int *host_data, int n, int **device_data) {
    hipError_t err;

    // Allocate device memory
    err = hipMalloc((void **)device_data, n * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    // Copy data from host to device
    err = hipMemcpy(*device_data, host_data, n * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to device: %s\n", hipGetErrorString(err));
        fflush(stderr);
        hipFree(*device_data);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


__host__
int device_to_host_data(int *host_data, int n, int *device_data) {
    hipError_t err;

    // Copy data from device to host
    err = hipMemcpy(host_data, device_data, n * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data back to host: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


__host__
int post_launch_barrier_and_check(void) {
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA synchronization error: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


__global__
void kernel_reverse(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_half = n >> 1;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n_half; i += stride) {
        int temp = data[i];
        int opposite_idx = n - i - 1;
        data[i] = data[opposite_idx];
        data[opposite_idx] = temp;
    }
}

}
