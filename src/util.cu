#include "util.cuh"
#include <stdio.h>


__host__
int host_to_device_data(int *host_data, int n, int **device_data) {
    hipError_t err;

    // Allocate device memory
    err = hipMalloc((void **)device_data, n * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    // Copy data from host to device
    err = hipMemcpy(*device_data, host_data, n * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to device: %s\n", hipGetErrorString(err));
        fflush(stderr);
        hipFree(*device_data);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


__host__
int device_to_host_data(int *host_data, int n, int *device_data) {
    hipError_t err;

    // Copy data from device to host
    err = hipMemcpy(host_data, device_data, n * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data back to host: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


__host__
int post_launch_barrier_and_check(void) {
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA synchronization error: %s\n", hipGetErrorString(err));
        fflush(stderr);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
